#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <cudaProfiler.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include "argmax_cuda.h"


__global__ void argmax_cuda(float *bisenet_output, float *argmax_output)
{

}

void argmax(float *bisenet_output, float *argmax_output)
{
    argmax_cuda<<<1,1>>>(bisenet_output, argmax_output);
}