#include "hip/hip_runtime.h"
#include "kernel_launch.h"
#include <math.h>
#include <bits/stdc++.h>


#define MIN_X_POINTCLOUD_RANGE 0
#define MAX_X_POINTCLOUD_RANGE 50
#define MIN_Y_POINTCLOUD_RANGE -25
#define MAX_Y_POINTCLOUD_RANGE 25

__constant__ float projection_matrix[16];

__global__ void painting_kernel(float *pointcloud, unsigned char *semantic_map, unsigned char* pointcloud_semantic, int n_points)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid >= n_points)
        return;

    float x = pointcloud[tid * POINTCLOUD_CHANNELS + 0];
    float y = pointcloud[tid * POINTCLOUD_CHANNELS + 1];
    float z = pointcloud[tid * POINTCLOUD_CHANNELS + 2];

    if (x < MIN_X_POINTCLOUD_RANGE || x > MAX_X_POINTCLOUD_RANGE || y < MIN_Y_POINTCLOUD_RANGE || y > MAX_Y_POINTCLOUD_RANGE)
        return;

    float projected_point[4] = {0};

    // transform each point with the projection matrix (proj is the dot product of the 3 matrices : P2 @ rect @ velo_to_cam)
    for (int i = 0; i < 4; i++)
    {
        float dotProduct = 0;
        dotProduct += projection_matrix[i * 4 + 0] * x;
        dotProduct += projection_matrix[i * 4 + 1] * y;
        dotProduct += projection_matrix[i * 4 + 2] * z;
        dotProduct += projection_matrix[i * 4 + 3]; // W =1 , * 1

        projected_point[i] = dotProduct;        
    }

    // devide by homogenious part (devide by z)
    projected_point[0] /= projected_point[2];
    projected_point[1] /= projected_point[2];

    // get x,y coordinates of the semantic map
    int y_semantic = (int)projected_point[0]; 
    int x_semantic = (int)projected_point[1];

    // only assign a label to the point if its projected point lies inside the semantic map, otherwise it is already has unlabeled value(255)
    if (x_semantic >= 0 && y_semantic >= 0 && x_semantic < HEIGHT_SEMANTIC_KITTI && y_semantic < WIDTH_SEMANTIC_KITTI)
        // assign a label to the point with the corresponding x,y projected point in the semantic map
        pointcloud_semantic[tid] = semantic_map[x_semantic * WIDTH_SEMANTIC_KITTI + y_semantic];
}

void pointpainting(float *pointcloud, unsigned char *semantic_map, float *proj_matrix, int n_points, 
                    unsigned char *pointcloud_semantic, hipStream_t stream)
{
    // set the constant projection matrix
    hipMemcpyToSymbol(HIP_SYMBOL(projection_matrix), proj_matrix, 16 * sizeof(float), 0UL, hipMemcpyHostToDevice);
    
    // set all points to be unlabeled till we label them
    hipMemset(pointcloud_semantic, UNLABELED_POINT, n_points * sizeof(unsigned char));

    // device multiprocessors
    // CUdevprop *properties;
    // cuDeviceGetProperties(properties, hipDeviceGet());

    int threadsPerBlock = 128;
    int numBlocks = ceil(double(n_points) / threadsPerBlock);
    std::cout << "pointpainting kernel with blocks = " << numBlocks << " & threads = " << threadsPerBlock << std::endl;
    painting_kernel<<<numBlocks, threadsPerBlock, 0, stream>>>(pointcloud, semantic_map, pointcloud_semantic, n_points);

    std::cout << "Painted called " << std::endl;
}